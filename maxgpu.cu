#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define BLOCKS 1024
#define THREADS 256

void getmaxcu(long *, long *);

/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/


int main(int argc, char *argv[])
{
   long size = 0;  // The size of the array
   long i;  // loop index
   long * numbers; // host copy of numbers array
   long * result; // host copy of result
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (long *)malloc(size * sizeof(long));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %ld\n", size);
       exit(1);
    }    


        result = (long *)malloc(size * sizeof(long));


    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand() % size;    
   

    // (1) Transfer numbers array
    long * num_d; //device copy of numbers array

    hipMalloc((void **) &num_d, size);
    hipMemcpy(num_d, numbers, size, hipMemcpyHostToDevice);

    long * result_d; //device copy of result

    // (2) Allocate device memory for result array
    hipMalloc((void **) &result_d, size);



     //(3) kernel launch code
    getmaxcu<<<BLOCKS,THREADS>>>(num_d, result_d);


     //(4) copy get max array from the device memory 
    hipMemcpy(result, result_d, size, hipMemcpyDeviceToHost);
    //free device memory
    hipFree(result_d);
    hipFree(num_d);

    free(numbers);
    free(result);
    exit(0);
}

__global__  void
getmaxcu(long * num_d, long * result_d)
{

  __shared__ long maxResult[THREADS * 2];
  int tx = threadIdx.x;

  for (int stride = THREADS*2; stride > 0; stride = stride /2 ) {
    __syncthreads();

    if (num_d[tx*2] > num_d[(tx*2)+1]) {

      num_d[tx*2] = maxResult[tx];

    }
    else {

      num_d[(tx*2)+1] = maxResult[tx];

    }


  }

  result_d[blockIdx.x] = maxResult[0];

}



