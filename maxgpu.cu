#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <iostream>

#define THREADS 1024

__global__  
void getmaxcu(unsigned int * numbers, unsigned int * result, unsigned int size)
{
  extern __shared__ unsigned int arr[];

  int tx = threadIdx.x;
  arr[tx] = numbers[tx];

  for (int stride = blockDim.x/2; stride > 0; stride = stride /2 ) {
    __syncthreads();
    if(tx<stride){
      if (arr[tx] < arr[tx+stride]) {
        arr[tx] = arr[tx]+stride;
      }
      __syncthreads();
    }
  }
  if(!tx){
    atomicMax(result, arr[0]);
  }
}
int main(int argc, char *argv[])
{
   unsigned int size = 0;  // The size of the array
   unsigned int i;  // loop index
   unsigned int * numbers; // host copy of numbers array
   unsigned int * result; // host copy of result
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);
    unsigned int grid=ceil((float)size/THREADS);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %ld\n", size);
       exit(1);
    }    


    result = (unsigned int *)malloc(size * sizeof(unsigned int));


    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand() % size;    
  
    unsigned int * device_numbers; 

    hipMalloc((void **) &device_numbers, sizeof(unsigned int)*size);
    hipMemcpy(device_numbers, numbers, sizeof(unsigned int) * size, hipMemcpyHostToDevice);

    unsigned int * device_result; 

    hipMalloc((void **) &device_result, size);

  dim3 dimGrid(grid);
  dim3 dimBlock(THREADS);
 

    getmaxcu<<<dimGrid,dimBlock,THREADS*sizeof(unsigned int)>>>(device_numbers, device_result, size);

    hipMemcpy(result, device_result, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipFree(device_result);
    hipFree(device_numbers);
    free(numbers);
    free(result);
    exit(0);
}





