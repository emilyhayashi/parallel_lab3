#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

unsigned int getmax(unsigned int *, unsigned int);

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;    
   
    printf(" The maximum number in the array is: %u\n", 
           getmax(numbers, size));

    free(numbers);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/

__global__
getmaxcu(long * num_arr)
{
    __shared__ long maxResult[THREADS * 2];
    int tx = threadIdx.x;

    for (int stride= THREADS * 2; stride > 0; stride = stride/2){
        __syncthreads();

        if (num_arr[tx*2] > num_d[(tx*2)+ 1]){
            num_arr[(tx*2)+1] = maxResult[tx];
        }
        else{
            num_arr[(tx*2) +1] = maxResult[tx];
        }
    }
    result[blockIdx.x] = maxResult[0];
}
