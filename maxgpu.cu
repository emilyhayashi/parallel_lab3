#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define BLOCKS 1024
#define THREADS 256

__global__  
void getmaxcu(long * num_d, long * result_d)
{

hipError_t hipGetDeviceProperties ( struct hipDeviceProp_t *   prop,
int   device   
);

printf(prop.maxThreadsPerBlock + "\n");


  __shared__ long maxResult[THREADS * 2];
  int tx = threadIdx.x;

  for (int stride = THREADS*2; stride > 0; stride = stride /2 ) {
    __syncthreads();

    if (num_d[tx*2] > num_d[(tx*2)+1]) {
      num_d[tx*2] = maxResult[tx];
    }
    else {
      num_d[(tx*2)+1] = maxResult[tx];
    }
  }
  result_d[blockIdx.x] = maxResult[0];
}

int main(int argc, char *argv[])
{
   long size = 0;  // The size of the array
   long i;  // loop index
   long * numbers; // host copy of numbers array
   long * result; // host copy of result
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (long *)malloc(size * sizeof(long));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %ld\n", size);
       exit(1);
    }    


        result = (long *)malloc(size * sizeof(long));


    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand() % size;    
  
    long * num_d; 

    hipMalloc((void **) &num_d, size);
    hipMemcpy(num_d, numbers, size, hipMemcpyHostToDevice);

    long * result_d; 

    hipMalloc((void **) &result_d, size);


    clock_t start, end;
    double cpu_time_used;
    start = clock();  
   
    getmaxcu<<<BLOCKS,THREADS>>>(num_d, result_d);
    end = clock();
    cpu_time_used = ((double) (end-start))/CLOCKS_PER_SEC;

    printf(" time taken %d\n", 
           cpu_time_used);


    hipMemcpy(result, result_d, size, hipMemcpyDeviceToHost);
    printf(" The maximum number in the array is: %u\n", 
           result);

    hipFree(result_d);
    hipFree(num_d);

    free(numbers);
    free(result);
    exit(0);
}





